// - Build
// nvcc -o "CUDA 기본/CudaNormal" "CUDA 기본/CudaNormal.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./CUDA 기본/CudaNormal"

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void Kernel(void) {
	printf("Hello, GPU World!\n");
}

int main() {
    Kernel<<<1, 4>>>();
	hipDeviceSynchronize();
}