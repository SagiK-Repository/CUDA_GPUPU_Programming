// - Build
// nvcc -o "01 CUDA 기본/04 CUDA ErrorCheck" "01 CUDA 기본/04 CUDA ErrorCheck.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./01 CUDA 기본/04 CUDA ErrorCheck"

#include<stdio.h>
#include "hip/hip_runtime.h"


#define CUDA_CHECK(val) { \
    if (val != hipSuccess) { \
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(val), __LINE__, __FILE__); \
        exit(1); \
    } \
}

__global__ void kernel(int* a) {
	*a+=5;
}

int main() {
    int host[3] = {10};
	int* device;

	CUDA_CHECK(hipMalloc((void**)&device, sizeof(int) * 10000000000000));

	hipFree(device);
}