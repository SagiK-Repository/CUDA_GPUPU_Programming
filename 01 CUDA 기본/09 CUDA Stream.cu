// - Build
// nvcc -o "01 CUDA 기본/09 CUDA Stream"  "01 CUDA 기본/09 CUDA Stream.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./01 CUDA 기본/09 CUDA Stream"

#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

__global__ void processDataKernel(float* data, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        data[i] = data[i] * data[i];
    }
}

void processWithoutStreams(float* h_data, int N) {
    float *d_data;

    hipMalloc((void**)&d_data, N * sizeof(float));
    hipMemcpy(d_data, h_data, N * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    processDataKernel<<<numBlocks, blockSize>>>(d_data, N);

    hipMemcpy(h_data, d_data, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_data);
}

void processWithStreams(float* h_data, int N) {
    hipStream_t stream;
    hipStreamCreate(&stream);

    float *d_data;

    hipMalloc((void**)&d_data, N * sizeof(float));
    hipMemcpyAsync(d_data, h_data, N * sizeof(float), hipMemcpyHostToDevice, stream); // Async

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    processDataKernel<<<numBlocks, blockSize, 0, stream>>>(d_data, N);

    hipMemcpyAsync(h_data, d_data, N * sizeof(float), hipMemcpyDeviceToHost, stream); // Async

    hipStreamSynchronize(stream);

    hipFree(d_data);
    hipStreamDestroy(stream);
}

int main() {
    const int N = 1 << 20; // 1M elements
    float *h_data;

    h_data = (float*)malloc(N * sizeof(float));

    for (int i = 0; i < N; ++i)
        h_data[i] = static_cast<float>(i); // initial

    // without stream
    auto start = std::chrono::high_resolution_clock::now();
    processWithoutStreams(h_data, N);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration_without_stream = end - start;

    // with stream
    start = std::chrono::high_resolution_clock::now();
    processWithStreams(h_data, N);
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration_with_stream = end - start;

    std::cout << "without stream: " << duration_without_stream.count() * 1000 << "ms" << std::endl;
    std::cout << "with stream: " << duration_with_stream.count() * 1000 << "ms" << std::endl;

    for (int i = 0; i < 10; ++i)
        std::cout << h_data[i] << " ";
    std::cout << std::endl;

    free(h_data);

    return 0;
}