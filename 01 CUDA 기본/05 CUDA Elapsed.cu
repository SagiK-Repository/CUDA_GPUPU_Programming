// - Build
// nvcc -o "01 CUDA 기본/05 CUDA Elapsed" "01 CUDA 기본/05 CUDA Elapsed.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./01 CUDA 기본/05 CUDA Elapsed"

#include<stdio.h>
#include "hip/hip_runtime.h"


#define CUDA_CHECK(val) { \
    if (val != hipSuccess) { \
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(val), __LINE__, __FILE__); \
        exit(1); \
    } \
}

__global__ void kernel(int* a) {
	*a+=5;
}

int main() {
    int host = 10;
	int* device;

	hipEvent_t start, stop;
	CUDA_CHECK(hipEventCreate(&start));
	CUDA_CHECK(hipEventCreate(&stop));

	CUDA_CHECK(hipMalloc((void**)&device, sizeof(int)));
	hipMemcpy(device, &host, sizeof(int), hipMemcpyHostToDevice);
	
	printf("\nbefore host : %d\n", host);

	CUDA_CHECK(hipEventRecord(start));
	kernel <<<2,2>>> (device);
	hipDeviceSynchronize();
	CUDA_CHECK(hipEventRecord(stop));

	hipMemcpy(&host, device, sizeof(int), hipMemcpyDeviceToHost);

	float msec_time = 0;
	CUDA_CHECK(hipEventElapsedTime(&msec_time, start, stop));
	printf("\nTime = %.3fms\n", msec_time);

	printf("\nafter host : %d\n", host);

	CUDA_CHECK(hipEventDestroy(start));
	CUDA_CHECK(hipEventDestroy(stop));

	hipFree(device);
}