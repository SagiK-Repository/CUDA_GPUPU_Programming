// - Build
// nvcc -o "01 CUDA 기본/02 CUDA Thread" "01 CUDA 기본/02 CUDA Thread.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./01 CUDA 기본/02 CUDA Thread"

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel(void) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	printf("\nGPU_Thread_num[%d] = (%d, %d, %d)", i, blockIdx.x, blockDim.x, threadIdx.x);
}

int main() {
	dim3 dimBlock(2, 1, 1);
	kernel <<<3,dimBlock>>>();
	hipDeviceSynchronize();
	printf("\ndimBlock.x = %d\n\n", dimBlock.x);

	dim3 dimBlock2(2, 2, 1);
	kernel <<<3,dimBlock2>>>();
	hipDeviceSynchronize();
	printf("\ndimBlock.y = %d\n\n", dimBlock.y);
    
	dim3 dimBlock3(2, 2, 2);
	kernel <<<3,dimBlock3>>>();
	hipDeviceSynchronize();
	printf("\ndimBlock.z = %d\n\n", dimBlock.z);
}