// - Build
// nvcc -o "01 CUDA 기본/01 CUDA Printf" "01 CUDA 기본/01 CUDA Printf.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./01 CUDA 기본/01 CUDA Printf"

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void Kernel(void) {
	printf("Hello, GPU World!\n");
}

int main() {
    Kernel<<<1, 4>>>();
	hipDeviceSynchronize();
}