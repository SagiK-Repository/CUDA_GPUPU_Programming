// - Build
// nvcc -o "01 CUDA 기본/06 CUDA Synchronize"  "01 CUDA 기본/06 CUDA Synchronize.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./01 CUDA 기본/06 CUDA Synchronize"

#include<stdio.h>
#include "hip/hip_runtime.h"


__global__ void kernel() {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ int a;
	int b=0;
	a+= i; b+= i;
	printf("S[%d][%d] : (%d, %d)\n", blockIdx.x, threadIdx.x, a, b);
	__syncthreads();
	printf("E[%d][%d] : (%d, %d)\n", blockIdx.x, threadIdx.x, a, b);
}

int main() {
	kernel <<<3,3>>> ();
	hipDeviceSynchronize();
}