// - Build
// nvcc -o "01 CUDA 기본/03 CUDA Memory" "01 CUDA 기본/03 CUDA Memory.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./01 CUDA 기본/03 CUDA Memory"

#include<stdio.h>
#include "hip/hip_runtime.h"


__global__ void kernel(int* a) {
	*a+=5;
}

int main() {
    int host = 10;
	int* device;

	hipMalloc((void**)&device, sizeof(int));
	hipMemcpy(device, &host, sizeof(int), hipMemcpyHostToDevice);
	
	printf("\nbefore host : %d\n", host);

	kernel <<<1,1>>> (device);
	hipDeviceSynchronize();

	hipMemcpy(&host, device, sizeof(int), hipMemcpyDeviceToHost);

	printf("\nafter host : %d\n", host);

	hipFree(device);
}