// - Build
// nvcc -o "01 CUDA 기본/08 CUDA Pinned Memory"  "01 CUDA 기본/08 CUDA Pinned Memory.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./01 CUDA 기본/08 CUDA Pinned Memory"

#include<stdio.h>
#include "hip/hip_runtime.h"


#define CUDA_CHECK(val) { \
    if (val != hipSuccess) { \
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(val), __LINE__, __FILE__); \
        exit(1); \
    } \
}

__global__ void kernel(int* a) {
	*a+=5;
}

int main() {
    int host = 10;
	
    //TimeCheck
	hipEvent_t start, stop, start0, stop0;
	CUDA_CHECK(hipEventCreate(&start));
	CUDA_CHECK(hipEventCreate(&stop));
	CUDA_CHECK(hipEventCreate(&start0));
	CUDA_CHECK(hipEventCreate(&stop0));
	
	int* device;
	
	CUDA_CHECK(hipEventRecord(start));
	
	CUDA_CHECK(hipMalloc((void**)&device, sizeof(int)));
	CUDA_CHECK(hipMemcpy(device, &host, sizeof(int), hipMemcpyHostToDevice));
	printf("before host : %d\n", host);
	kernel <<<2, 2>>> (device);
	hipDeviceSynchronize();
	CUDA_CHECK(hipMemcpy(&host, device, sizeof(int), hipMemcpyDeviceToHost));
	printf("after host : %d\n", host);
	
	CUDA_CHECK(hipEventRecord(stop));

	float msec_time = 0;
	hipDeviceSynchronize();
	CUDA_CHECK(hipEventElapsedTime(&msec_time, start, stop));
	printf("\nTime = %.3f ms\n\n\n", msec_time);

	// Pinned
	int* hosts;
	CUDA_CHECK(hipEventRecord(start0));
	
	CUDA_CHECK(hipHostMalloc((void**)&hosts, sizeof(int), hipHostMallocDefault, hipHostMallocDefault)); //Host Pinned

	hosts = &host;
	CUDA_CHECK(hipMalloc((void**)&device, sizeof(int)));
	CUDA_CHECK(hipMemcpy(device, hosts, sizeof(int), hipMemcpyHostToDevice));
	printf("before host : %d\n", *hosts);
	kernel <<<2, 2>>> (device);
	hipDeviceSynchronize();
	CUDA_CHECK(hipMemcpy(hosts, device, sizeof(int), hipMemcpyDeviceToHost));
	printf("after host : %d\n", *hosts);

	CUDA_CHECK(hipEventRecord(stop0));

	msec_time = 0;
	hipDeviceSynchronize();
	CUDA_CHECK(hipEventElapsedTime(&msec_time, start0, stop0));
	printf("\nTime = %.3f ms\n", msec_time);

	CUDA_CHECK(hipFree(device));
	CUDA_CHECK(hipEventDestroy(start));
	CUDA_CHECK(hipEventDestroy(stop));
	CUDA_CHECK(hipEventDestroy(start0));
	CUDA_CHECK(hipEventDestroy(stop0));

    return 0;
}