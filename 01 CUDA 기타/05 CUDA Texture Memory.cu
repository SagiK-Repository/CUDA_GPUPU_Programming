// - Build
// nvcc -o "01 CUDA 기타/05 CUDA Texture Memory"  "01 CUDA 기타/05 CUDA Texture Memory.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./01 CUDA 기타/05 CUDA Texture Memory"

#include <iostream>
#include <hip/hip_runtime.h>

hipTextureObject_t texObj; // 텍스처 객체 선언

__global__ void processImage(float *output, int width, int height, hipTextureObject_t texObj) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        float pixelValue = tex2D<float>(texObj, x, y); // read pixel from texture memory
        output[y * width + x] = pixelValue * 2.0f; // pixel * 2
    }
}

int main() {
    const int width = 512;
    const int height = 512;
    size_t size = width * height * sizeof(float);

    float *h_image = (float*)malloc(size);
    for (int i = 0; i < width * height; ++i) {
        h_image[i] = static_cast<float>(i % 256); // sample pattern
    }

    float *d_output;
    hipMalloc((void**)&d_output, size);

    // Set 2D Array Texture Memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray *d_textureArray;
    hipMallocArray(&d_textureArray, &channelDesc, width, height);

    // 2D 배열로 데이터 복사
    hipMemcpy2DToArray(d_textureArray, 0, 0, h_image, width * sizeof(float), width * sizeof(float), height, hipMemcpyHostToDevice);

    // 텍스처 객체 생성
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = d_textureArray;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    processImage<<<gridSize, blockSize>>>(d_output, width, height, texObj);
    hipDeviceSynchronize();

    float *h_output = (float*)malloc(size);
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    std::cout << "Processed image (first 10 pixels): ";
    for (int i = 0; i < 10; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    hipDestroyTextureObject(texObj);
    hipFree(d_output);
    hipFreeArray(d_textureArray);
    free(h_image);
    free(h_output);

    return 0;
}
