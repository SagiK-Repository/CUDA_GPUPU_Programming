#include "hip/hip_runtime.h"
// - Build
// nvcc -o "01 CUDA 기타/05 CUDA Texture Memory"  "01 CUDA 기타/05 CUDA Texture Memory.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./01 CUDA 기타/05 CUDA Texture Memory"

#include <iostream>
#include <hip/hip_runtime.h>

texture<float, hipTextureType2D, hipReadModeElementType> texRef;

__global__ void processImage(float *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        float pixelValue = tex2D(texRef, x, y); // read pixel from texture memory
        output[y * width + x] = pixelValue * 2.0f; // pixel * 2
    }
}

int main() {
    const int width = 512;
    const int height = 512;
    size_t size = width * height * sizeof(float);

    float *h_image = (float*)malloc(size);
    for (int i = 0; i < width * height; ++i) {
        h_image[i] = static_cast<float>(i % 256); // sample pattern
    }

    float *d_output;
    hipMalloc((void**)&d_output, size);

    // Set 2D Array Texture Memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray *d_textureArray;
    hipMallocArray(&d_textureArray, &channelDesc, width, height);
    hipMemcpyToArray(d_textureArray, 0, 0, h_image, size, hipMemcpyHostToDevice);

    // Binding
    hipBindTextureToArray(texRef, d_textureArray);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    processImage<<<gridSize, blockSize>>>(d_output, width, height);
    hipDeviceSynchronize();

    float *h_output = (float*)malloc(size);
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    std::cout << "Processed image (first 10 pixels): ";
    for (int i = 0; i < 10; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_output);
    hipFreeArray(d_textureArray);
    free(h_image);
    free(h_output);

    return 0;
}