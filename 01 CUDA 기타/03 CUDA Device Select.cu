// - Build
// nvcc -o "01 CUDA 기타/03 CUDA Device Select"  "01 CUDA 기타/03 CUDA Device Select.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./01 CUDA 기타/03 CUDA Device Select"

#include <iostream>
#include <hip/hip_runtime.h>


__global__ void square(int *d_data, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        d_data[idx] = d_data[idx] * d_data[idx];
    }
}

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    
    if (deviceCount == 0) {
        std::cerr << "No CUDA devices available." << std::endl;
        return -1;
    }
    
    std::cout << "Number of CUDA devices: " << deviceCount << std::endl;

    int device = 0;
    hipSetDevice(device);

    // Device Info
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    std::cout << "Using Device: " << prop.name << std::endl;

    const int N = 512;
    size_t size = N * sizeof(int);

    int *h_data = (int*)malloc(size);
    for (int i = 0; i < N; ++i) {
        h_data[i] = i;
    }

    int *d_data;
    hipMalloc((void**)&d_data, size);
    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    square<<<numBlocks, blockSize>>>(d_data, N);

    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_data);
    free(h_data);

    return 0;
}