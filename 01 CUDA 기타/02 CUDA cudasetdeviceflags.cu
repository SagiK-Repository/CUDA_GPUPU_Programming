// - Build
// nvcc -o "01 CUDA 기타/02 CUDA cudasetdeviceflags"  "01 CUDA 기타/02 CUDA cudasetdeviceflags.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./01 CUDA 기타/02 CUDA cudasetdeviceflags"

#include <iostream>
#include <hip/hip_runtime.h>

__global__ void square(int *d_data, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        d_data[idx] = d_data[idx] * d_data[idx];
    }
}

int main() {
    const int N = 512;
    size_t size = N * sizeof(int);

    hipError_t err = hipSetDeviceFlags(hipDeviceMapHost);
    if (err != hipSuccess) {
        std::cerr << "Error setting device flags: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Mapped Memory
    int *h_data;
    hipHostAlloc((void**)&h_data, size, hipHostMallocDefault);

    for (int i = 0; i < N; ++i) {
        h_data[i] = i;
    }

    // Device Memory Pointer
    int *d_data;
    hipHostGetDevicePointer(&d_data, h_data, 0);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    square<<<numBlocks, blockSize>>>(d_data, N);
    
    hipDeviceSynchronize();

    for (int i = 0; i < N; ++i) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    hipHostFree(h_data);

    return 0;
}