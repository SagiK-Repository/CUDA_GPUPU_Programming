// - Build
// nvcc -o "01 CUDA 기타/01 CUDA Mapped Memory"  "01 CUDA 기타/01 CUDA Mapped Memory.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./01 CUDA 기타/01 CUDA Mapped Memory"

#include <iostream>
#include <hip/hip_runtime.h>

__global__ void square(int *d_data, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        d_data[idx] = d_data[idx] * d_data[idx];
    }
}

int main() {
    const int N = 512;
    size_t size = N * sizeof(int);

    // Mapped Memory
    int *h_data;
    hipHostAlloc((void**)&h_data, size, hipHostMallocMapped);

    for (int i = 0; i < N; ++i)
        h_data[i] = i;

    // Device Memory Pointer
    int *d_data;
    hipHostGetDevicePointer(&d_data, h_data, 0);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    square<<<numBlocks, blockSize>>>(d_data, N);

    for (int i = 0; i < N; ++i)
        std::cout << h_data[i] << " ";
    std::cout << std::endl;

    hipHostFree(h_data);
    return 0;
}