#include "hip/hip_runtime.h"
// - Build
// nvcc -o "01 CUDA 기타/06 CUDA Layerd Texture"  "01 CUDA 기타/06 CUDA Layerd Texture.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./01 CUDA 기타/06 CUDA Layerd Texture"

#include <iostream>
#include <hip/hip_runtime.h>
#include ""

texture<float, cudaTextureType2DArray, hipReadModeElementType> texRef;

__global__ void processLayeredTexture(float *output, int width, int height, int layer) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        float pixelValue = tex2DLayered(texRef, x, y, layer);  // read pixel from texture memory
        output[y * width + x] = pixelValue * 2.0f; // pixel * 2
    }
}

int main() {
    const int width = 512;
    const int height = 512;
    const int layers = 3;
    size_t size = width * height * sizeof(float);

    float *h_layers = (float*)malloc(size * layers);
    for (int layer = 0; layer < layers; ++layer) {
        for (int i = 0; i < width * height; ++i) {
            h_layers[layer * width * height + i] = static_cast<float>(i % 256) + layer; // 레이어별로 다른 패턴 생성
        }
    }

    float *d_output;
    hipMalloc((void**)&d_output, size);

    // Set 2D Array Texture Memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray *d_textureArray;
    hipMallocArray(&d_textureArray, &channelDesc, width, height, layers);

    // Copy Host Memory to Texture Array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(h_layers, width * sizeof(float), width, height * layers);
    copyParams.dstArray = d_textureArray;
    copyParams.extent = make_hipExtent(width, height, layers);
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);

    // Binding
    hipBindTextureToArray(texRef, d_textureArray);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    processLayeredTexture<<<gridSize, blockSize>>>(d_output, width, height, 0); // First Layer
    hipDeviceSynchronize();

    float *h_output = (float*)malloc(size);
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    std::cout << "Processed image from layer 0 (first 10 pixels): ";
    for (int i = 0; i < 10; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_output);
    hipFreeArray(d_textureArray);
    free(h_layers);
    free(h_output);

    return 0;
}
