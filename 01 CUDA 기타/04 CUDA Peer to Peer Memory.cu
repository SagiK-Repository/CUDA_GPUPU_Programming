// - Build
// nvcc -o "01 CUDA 기타/04 CUDA Peer to Peer Memory"  "01 CUDA 기타/04 CUDA Peer to Peer Memory.cu" --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// - Run
// "./01 CUDA 기타/04 CUDA Peer to Peer Memory"

#include <iostream>
#include <hip/hip_runtime.h>

__global__ void initializeArray(int *data, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        data[idx] = idx;
    }
}

void printArray(int *data, int N) {
    for (int i = 0; i < N; ++i) {
        std::cout << data[i] << " ";
    }
    std::cout << std::endl;
}

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount < 2) {
        std::cerr << "At least two CUDA devices are required for this example." << std::endl;
        return -1;
    }

    // GPU 0
    int N = 512;
    size_t size = N * sizeof(int);
    int *d_data0, *d_data1;

    // GPU 0
    hipSetDevice(0);
    hipMalloc((void**)&d_data0, size);
    
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    initializeArray<<<numBlocks, blockSize>>>(d_data0, N);
    hipDeviceSynchronize();

    // GPU 1
    hipSetDevice(1);
    hipMalloc((void**)&d_data1, size);

    // P2P Memory
    hipDeviceEnablePeerAccess(0, 0); // GPU 1 Access to GPU 0
    hipDeviceEnablePeerAccess(1, 0); // GPU 0 Access to GPU 1

    // GPU 0, Copy to GPU 1
    hipSetDevice(0);
    hipMemcpyPeer(d_data1, 1, d_data0, 0, size); // Copy GPU 0 d_data0 -> GPU 1 d_data1

    hipSetDevice(1);
    int *h_data1 = (int*)malloc(size);
    hipMemcpy(h_data1, d_data1, size, hipMemcpyDeviceToHost);
    
    std::cout << "Data in GPU 1 after copying from GPU 0: ";
    printArray(h_data1, N);

    hipSetDevice(0);
    hipFree(d_data0);
    hipSetDevice(1);
    hipFree(d_data1);
    free(h_data1);

    return 0;
}